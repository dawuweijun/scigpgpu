#include "hip/hip_runtime.h"
/*
* Scilab ( http://www.scilab.org/ ) - This file is part of Scilab
* Copyright (C) DIGITEO - 2011 - Cedric DELAMARRE
*
* This file must be used under the terms of the CeCILL.
* This source file is licensed as described in the file COPYING, which
* you should have received as part of this distribution.  The terms
* are also available at
* http://www.cecill.info/licences/Licence_CeCILL_V2-en.txt
*
*/

#include <math.h>
#include "idmax.h"

__global__ void idmax_kernel(double* d, int elems, double* result)
{
	int posInGrid  = blockIdx.x  * blockDim.x + threadIdx.x;
   	extern __shared__ double accumResult[];

	if(posInGrid < elems)
	{
		accumResult[threadIdx.x] = d[posInGrid];

		for(int i = posInGrid+blockDim.x*gridDim.x; i < elems; i += blockDim.x*gridDim.x)
		{
				if(d[i] > accumResult[threadIdx.x])
					accumResult[threadIdx.x] = d[i];
		}
		__syncthreads();

		for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
		{
			if(threadIdx.x < stride && posInGrid+stride < elems)
			{
				if(accumResult[stride + threadIdx.x] > accumResult[threadIdx.x])
					accumResult[threadIdx.x] = accumResult[stride + threadIdx.x];
			}
			__syncthreads();
		}

		if(threadIdx.x == 0) result[blockIdx.x] = accumResult[0];
	}
}

hipError_t cudaIdmax(int elems, double* d, double* res)
{
	double* input  	= NULL;
	double* output 	= NULL;
	hipError_t cudaStat = hipGetLastError();

	try
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess) throw cudaStat;

		int threadMax 	= deviceProp.maxThreadsDim[0];
		int blockMax  	= deviceProp.maxGridSize[0];

		int dimgrid  	= (int) ceil((float)elems/threadMax);

		input = d;

		if(blockMax < dimgrid)
			dimgrid = blockMax;

		while(true)
		{
			hipMalloc((void**)&output,dimgrid*sizeof(double));
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			dim3 block(threadMax, 1, 1);
		   	dim3 grid(dimgrid, 1, 1);
		   	idmax_kernel<<<grid, block, threadMax*8>>>(input,elems,output);

            cudaStat = hipGetLastError();
            if (cudaStat != hipSuccess) throw cudaStat;

            cudaStat = hipDeviceSynchronize();
			if (cudaStat != hipSuccess) throw cudaStat;

			if(dimgrid == 1)
				break;

			elems = dimgrid;

			if(input != d)
				hipFree(input);

			hipMalloc((void**)&input,elems*sizeof(double));
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			hipMemcpy(input,output,elems*sizeof(double),hipMemcpyDeviceToDevice);
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			hipFree(output);
			dimgrid = (int) ceil((float)elems/threadMax);
		}

		hipMemcpy(res,output,sizeof(double),hipMemcpyDeviceToHost);
		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess) throw cudaStat;

		hipFree(output);
		if(input != d)
			hipFree(input);

		return hipSuccess;
	}
	catch(hipError_t cudaE)
	{
		if(input != NULL && input != d) hipFree(input);
		if(output != NULL) hipFree(output);
		return cudaE;
	}
}
