#include "hip/hip_runtime.h"
/*
* Scilab ( http://www.scilab.org/ ) - This file is part of Scilab
* Copyright (C) DIGITEO - 2011 - Cedric DELAMARRE
*
* This file must be used under the terms of the CeCILL.
* This source file is licensed as described in the file COPYING, which
* you should have received as part of this distribution.  The terms
* are also available at
* http://www.cecill.info/licences/Licence_CeCILL_V2-en.txt
*
*/

#include <math.h>
#include <stdio.h>
#include <hip/hip_complex.h>
#include "zsum.h"

__global__ void zsum_kernel(hipDoubleComplex* d, int elems, hipDoubleComplex* result)
{
	int posInGrid  = blockIdx.x  * blockDim.x + threadIdx.x;
   	extern __shared__ hipDoubleComplex accumResult[];

	accumResult[threadIdx.x].x = 0.0;
	accumResult[threadIdx.x].y = 0.0;

	for(int i = posInGrid; i < elems; i += blockDim.x*gridDim.x)
	{
		accumResult[threadIdx.x].x += d[i].x;
		accumResult[threadIdx.x].y += d[i].y;
	}
	__syncthreads();

	for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
	{
		if(threadIdx.x < stride)
		{
			accumResult[threadIdx.x].x += accumResult[stride + threadIdx.x].x;
			accumResult[threadIdx.x].y += accumResult[stride + threadIdx.x].y;
		}
		__syncthreads();
	}

	if(threadIdx.x == 0)
	{
		result[blockIdx.x].x = accumResult[0].x;
		result[blockIdx.x].y = accumResult[0].y;
	}
}

hipError_t cudaZsum(int elems, hipDoubleComplex* d, hipDoubleComplex* res)
{
	hipDoubleComplex* input	= NULL;
	hipDoubleComplex* output	= NULL;
	hipError_t cudaStat = hipGetLastError();

	try
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess) throw cudaStat;

		int threadMax = deviceProp.maxThreadsDim[0];
		int blockMax  = deviceProp.maxGridSize[0];

		int dimgrid  	= (int) ceil((float)elems/threadMax);

		input = d;

		if(blockMax < dimgrid)
			dimgrid = blockMax;

		while(true)
		{
			hipMalloc((void**)&output,dimgrid*sizeof(hipDoubleComplex));
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			dim3 block(threadMax, 1, 1);
		   	dim3 grid(dimgrid, 1, 1);
		   	zsum_kernel<<<grid, block, threadMax*16>>>(input,elems,output);

            cudaStat = hipGetLastError();
            if (cudaStat != hipSuccess) throw cudaStat;

            cudaStat = hipDeviceSynchronize();
			if (cudaStat != hipSuccess) throw cudaStat;

			if(dimgrid == 1)
				break;

			elems = dimgrid;

			if(input != d)
				hipFree(input);

			hipMalloc((void**)&input,elems*sizeof(hipDoubleComplex));
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			hipMemcpy(input,output,elems*sizeof(hipDoubleComplex),hipMemcpyDeviceToDevice);
			cudaStat = hipGetLastError();
			if (cudaStat != hipSuccess) throw cudaStat;

			hipFree(output);
			dimgrid = (int) ceil((float)elems/threadMax);
		}

		hipMemcpy(res,output,sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
		cudaStat = hipGetLastError();
		if (cudaStat != hipSuccess) throw cudaStat;

		hipFree(output);
		if(input != d)
			hipFree(input);

		return hipSuccess;
	}
	catch(hipError_t cudaE)
	{
		if(input != NULL && input != d) hipFree(input);
		if(output != NULL) hipFree(output);
		return cudaE;
	}
}
